#include "hip/hip_runtime.h"
﻿#include <stdio.h>
#include <stdlib.h>

#define base_int unsigned int

#define NUM_ELEM 16
unsigned int data[NUM_ELEM] = { 124, 20, 5, 86, 240, 183, 68, 18, 24, 120, 15, 6, 40, 283, 38, 128 }

__device__ void radix_sort(base_int *arr, base_int n, base_int *tmp_1, base_int tid, base_int tdim)
{
	for (base_int bit = 0; bit < 32; bit++)
	{
		base_int base_0 = 0;
		base_int base_1 = 0;
		base_int bit_mask = (1 << bit);
		for (base_int i = 0; i < n; i += tdim)
		{
			base_int x = arr[i + tid];
			if ((x&bit_mask)>0)
			{
				tmp_1[base_1 + tid] = x;
				base_1 += tdim;
			}
			else
			{
				arr[base_0 + tid] = x;
				base_0 += tdim;
			}
		}
		for (base_int i = 0; i < base_1; i += tdim)
		{
			arr[i + base_0 + tid] = tmp_1[i + tid];
		}
	}
}

__device__ void merge_array2(base_int *arr, base_int *brr, base_int tid, base_int tdim, base_int n)
{
	base_int n1=n/tdim;
	__shared__ base_int list_index[1024];
	__shared__ base_int red_val[1024];
	__shared__ base_int red_idx[1024];
	list_index[tid] = 0;
	red_val[tid] = 0;
	red_idx[tid] = 0;
	__syncthreads();
	for (base_int i = 0; i < n; i++)
	{
		base_int tid_max = tdim >> 1;
		base_int x;
		if (list_index[tid] < n1)
		{
			base_int idx = tid + (list_index[tid] * tdim);
			x = arr[idx];
		}
		else
		{
			x = 0xFFFFFFFF;
		}
		red_val[tid] = x;
		red_idx[tid] = tid;
		__syncthreads();
		while (tid_max != 0)
		{
			if (tid < tid_max)
			{
				base_int idx1 = tid + tid_max;
				base_int vall = red_val[idx1];
				if (red_val[tid] > vall)
				{
					red_val[tid] = vall;
					red_idx[tid] = red_idx[idx1];
				}
			}
			tid_max >>= 1;
			__syncthreads;
		}
		if (tid == 0)
		{
			list_index[red_idx[0]]++;
			brr[i] = red_val[0];
		}

		__syncthreads(); 
	}
}	

__global__ void merge_sort_gpu(base_int *arr, base_int tdim, base_int n)
{
	base_int tid = blockDim.x * blockIdx.x + threadIdx.x;
	__shared__ base_int tmp[NUM_ELEM];
	__shared__ base_int tmp1[NUM_ELEM];

	for (base_int i = 0; i < n; i += tdim)
	{
		tmp[i + tid] = arr[i + tid];
	}
	__syncthreads();
	radix_sort(tmp, n, tmp1, tid, tdim);
	merge_array2(tmp, arr, tid, tdim, n);
}

int main(){

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	size_t size = NUM_ELEM*sizeof(int);
	base_int *d_a;

	hipMalloc((void **)&d_a, size);
	hipMemcpy(d_a, data, size, hipMemcpyHostToDevice);

	//stride
	base_int tdim = 2;
	hipEventRecord(start);
	merge_sort_gpu << <1, tdim >> >(d_a, tdim, NUM_ELEM);
	hipEventRecord(stop);

	hipMemcpy(data, d_a, size, hipMemcpyDeviceToHost);
	hipFree(d_a);

	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("Elapsed time (ms): %f\n", milliseconds);

	for (int i = 0; i < NUM_ELEM; i++)
	{
		printf("%d ", data[i]);
	}
	printf("\n");
		
	
	return EXIT_SUCCESS;
}

